#include "hip/hip_runtime.h"
// for linux:
// nvcc self-assembly-gpu.cu -o temp -lcudart

#include <sys/time.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

// ---- BODY PROPERTIES ---- //
#define N 6
#define MASS 1.0 // 0.0000000000131946891 //estimate with density 1.05g per cm cube
#define DIAMETER_PS 1.0 // Diameter of polystyrene spheres 1 micron
#define DIAMETER_NIPAM 0.08 // Diameter of polyNIPAM microgel particles 80 nanometers

// ---- FORCE FUNCTION ---- //
// Constant force for piecewise step function
#define MAX_ATTRACTION 10.0
#define REPULTION_MULTIPLIER 5.0
#define SHORT_RANGE_MULTIPLIER 0.01
#define LONG_RANGE_MULTIPLIER 0.5
#define LONG_RANGE_DISTANCE_MULTIPLIER 3.0
#define INITIAL_VELOCITY 0.5
#define DAMP .3
#define DT 0.0001

// ---- CONFIG PROPERTIES ---- //
// #define NUMBER_OF_RUNS 200
#define NUMBER_OF_THREADS 1 // this is the number of runs
#define MAX_TOTAL_KINETIC_ENERGY 0.002
#define INITIAL_SEPARATION 1.1
#define NO 0
#define YES 1

// ---- RESULTS PROPERTIES ---- //
float octa_count = 0.0;
float tetra_count = 0.0;
float other_count = 0.0;

double *OCTA_CPU, *TETRA_CPU, *OTHER_CPU;
double *OCTA_GPU, *TETRA_GPU, *OTHER_GPU;
float *RANDOM_POS_CPU, *RANDOM_VEL_CPU;
float *RANDOM_POS_GPU, *RANDOM_VEL_GPU;

// GPU setup
dim3 dimBlock;
dim3 dimGrid;

void set_up_cuda_devices()
{
    // number of threads per block will
    // always be maxed at 1024
    dimBlock.x = 1024;
    dimBlock.y = 1;
    dimBlock.z = 1;

    // figure out how many blocks you need
    if (NUMBER_OF_THREADS % dimBlock.x != 0) {
        dimGrid.x = NUMBER_OF_THREADS / dimBlock.x + 1;
    }
    else {
        dimGrid.x = NUMBER_OF_THREADS / dimBlock.x;
    }
    dimGrid.y = 1;
    dimGrid.z = 1;
    printf("Number of blocks: %i\n", dimGrid.x);
}

void allocate_memory()
{
    // Allocate Host (CPU) Memory with 0s
    OCTA_CPU = (double*)calloc(NUMBER_OF_THREADS, sizeof(double));
    TETRA_CPU = (double*)calloc(NUMBER_OF_THREADS, sizeof(double));
    OTHER_CPU = (double*)calloc(NUMBER_OF_THREADS, sizeof(double));
    RANDOM_POS_CPU = (float*)malloc(N*3*NUMBER_OF_THREADS*sizeof(float));
    RANDOM_VEL_CPU = (float*)malloc(N*3*NUMBER_OF_THREADS*sizeof(float));

    // generate random numbers for each experiment (flattened 3D array)
    // this includes random numbers for positions and velocities
	for (int i = 0; i < NUMBER_OF_THREADS; i++)
    {
        for (int j = 0; j < N*3; j++)
        {
            RANDOM_POS_CPU[N*3*i+j] = ((float)rand()/(float)RAND_MAX);
            RANDOM_VEL_CPU[N*3*i+j] = ((float)rand()/(float)RAND_MAX);
        }
    }

    // how to get forces for each body
    // for (int i=0; i < N*3*NUMBER_OF_THREADS; i+=3)
    // {
    //     printf("--------------------- random positions -----------------\n");
    //     printf("RANDOM_POS_CPU[%i]: %.15f\n", i, RANDOM_POS_CPU[i]);
    //     printf("RANDOM_POS_CPU[%i]: %.15f\n", i+1, RANDOM_POS_CPU[i+1]);
    //     printf("RANDOM_POS_CPU[%i]: %.15f\n", i+2, RANDOM_POS_CPU[i+2]);
    //     printf("--------------------- random velocities -----------------\n");
    //     printf("RANDOM_VEL_CPU[%i]: %.15f\n", i, RANDOM_VEL_CPU[i]);
    //     printf("RANDOM_VEL_CPU[%i]: %.15f\n", i+1, RANDOM_VEL_CPU[i+1]);
    //     printf("RANDOM_VEL_CPU[%i]: %.15f\n", i+2, RANDOM_VEL_CPU[i+2]);
    // }

    // Allocate Device (GPU) Memory and allocates the value of the specific pointer/array
    hipMalloc(&OCTA_GPU, NUMBER_OF_THREADS*sizeof(double));
    hipMalloc(&TETRA_GPU, NUMBER_OF_THREADS*sizeof(double));
    hipMalloc(&OTHER_GPU, dimGrid.x*dimBlock.x*sizeof(double));
    hipMalloc(&RANDOM_POS_GPU, N*3*NUMBER_OF_THREADS*sizeof(float));
    hipMalloc(&RANDOM_VEL_GPU, N*3*NUMBER_OF_THREADS*sizeof(float));
}

// Cleaning up memory on both host and device after we are finished.
void clean_up(double *OCTA_CPU, double *TETRA_CPU, double *OTHER_CPU, double *OCTA_GPU, double *TETRA_GPU, double *OTHER_GPU)
{
    free(OCTA_CPU); free(TETRA_CPU); free(OTHER_CPU);
    hipFree(OCTA_GPU); hipFree(TETRA_GPU); hipFree(OTHER_GPU);
}

// custom error function to make sure
// the GPU did what it was supposed to do
void errorCheck(const char *message)
{
  hipError_t  error;
  error = hipGetLastError();

  if(error != hipSuccess)
  {
    printf("\n CUDA ERROR: %s - %s\n", message, hipGetErrorString(error));
    exit(0);
  }
}

__global__ void self_assemble(double *OCTA_GPU, double *TETRA_GPU,
                              double *OTHER_GPU, float *RANDOM_POS_GPU,
                              float *RANDOM_VEL_GPU, int number_of_threads,
                              int n, unsigned int seed)
{
    // globals to store positions, velocities, and forces
    float p[6][3], v[6][3], f[6][3], mass[6];
    // OpenGL box size
    float xMin = -4.0;
    float xMax =  4.0;
    float yMin = -4.0;
    float yMax =  4.0;
    float zMin = -4.0;
    float zMax =  4.0;

    // each thread will perform an entire experiment
    int id = blockDim.x*blockIdx.x + threadIdx.x;
    if (id < number_of_threads) { // Make sure we do not go out of bounds
        // ------- SET INITIAL CONDITIONS ------- //
    	int i, j, ok_config;
    	float mag, distance, dx, dy, dz;
    	ok_config = NO;

    	while(ok_config == NO)
    	{
    		for (i = 0; i < n; i++)
    		{
    			// initialize mass of bodies
    			mass[i] = 1.0;
    			// intitialize positions
    			p[i][0] = RANDOM_POS_GPU[N*3*id+(i*3)] * (xMax - xMin) - ((xMax-xMin)/2);
    			p[i][1] = RANDOM_POS_GPU[N*3*id+(i*3+1)] * (yMax - yMin) - ((yMax-yMin)/2);
    			p[i][2] = RANDOM_POS_GPU[N*3*id+(i*3+2)] * (zMax - zMin) - ((yMax-yMin)/2);
    			// initialize velocities
    			mag = sqrt(p[i][0]*p[i][0]+p[i][1]*p[i][1]+p[i][2]*p[i][2]);
    			v[i][0] = INITIAL_VELOCITY*(-p[i][0]/mag)*RANDOM_VEL_GPU[N*3*id+(i*3)];
    			v[i][1] = INITIAL_VELOCITY*(-p[i][1]/mag)*RANDOM_VEL_GPU[N*3*id+(i*3+1)];
    			v[i][2] = INITIAL_VELOCITY*(-p[i][2]/mag)*RANDOM_VEL_GPU[N*3*id+(i*3+2)];
    		}
    		// make sure each body is a minimum distance from all the others
    		ok_config = YES;
    		for(i = 0; i < (n - 1); i++)
    		{
    			for(j = i + 1; j < n; j++)
    			{
    				dx = p[i][0]-p[j][0];
    				dy = p[i][1]-p[j][1];
    				dz = p[i][2]-p[j][2];
    				distance = sqrt(dx*dx + dy*dy + dz*dz);
    				// if(distance <= INITIAL_SEPARATION) {
    				// 	// printf("bodies too close!\n");
    				// 	ok_config = NO;
    				// }
    			}
    		}
    	}
        printf("initial positions:\n");
		for(i = 0; i < n; i++)
        {
            printf("p[%i][0]: %.15f\n", i, p[i][0]);
            printf("p[%i][1]: %.15f\n", i, p[i][1]);
            printf("p[%i][2]: %.15f\n", i, p[i][2]);
        }
        // -------------------------------------- //
		float total_kinetic_energy = 1.0;
        // printf("total kinetic energy: %.15f\n", total_kinetic_energy);
        // printf("DIAMETER_PS: %.15f\n", DIAMETER_PS);
		// stop updates when bodies have stopped moving
		int test = 0;
		while(total_kinetic_energy > MAX_TOTAL_KINETIC_ENERGY)
		{
            // ------- GET FORCES ------- //
        	float squared_distance;
        	float force_mag;

        	// initialize forces to 0
        	for (i = 0; i < n; i++)
        	{
        		f[i][0] = 0.0;
        		f[i][1] = 0.0;
        		f[i][2] = 0.0;
        	}
            // loop through every body
        	for (i = 0; i < n; i++)
        	{
        		// for each body, calculate distance and
        		// force from every other body
        		for (j = i+1; j < n; j++)
        		{
        			dx = p[j][0]-p[i][0];
        			dy = p[j][1]-p[i][1];
        			dz = p[j][2]-p[i][2];
                    // printf("p[%i][0]: %.15f\n", j, p[j][0]);
                    // printf("dx: %.15f\n", dx);
                    // printf("dy: %.15f\n", dy);
                    // printf("dz: %.15f\n", dz);
        			squared_distance = dx*dx + dy*dy + dz*dz;
        			distance = sqrt(squared_distance);
                    // printf("distance: %.15f\n", distance);

        			if (distance < DIAMETER_PS) // d < 1
        			{
        				force_mag = -REPULTION_MULTIPLIER*MAX_ATTRACTION; // -50
        			}
        			else if (distance < DIAMETER_PS + DIAMETER_NIPAM) // d < 1.08
        			{
        				force_mag =  MAX_ATTRACTION; //10
        			}
        			else if (distance < LONG_RANGE_DISTANCE_MULTIPLIER*DIAMETER_PS) // d < 3
        			{
        				force_mag =  MAX_ATTRACTION*SHORT_RANGE_MULTIPLIER; //.1
        			}
        			// make extra force that pulls to center
        			else force_mag = MAX_ATTRACTION*LONG_RANGE_MULTIPLIER; // 5
        			f[i][0] += force_mag*dx/distance;
        			f[j][0] -= force_mag*dx/distance;
        			f[i][1] += force_mag*dy/distance;
        			f[j][1] -= force_mag*dy/distance;
        			f[i][2] += force_mag*dz/distance;
        			f[j][2] -= force_mag*dz/distance;
        		}
        	}
            // printf("force mag: %.15f\n", force_mag);
    		// for(i = 0; i < n; i++)
            // {
            //     printf("f[%i][0]: %.15f\n", i, force_mag*dx/distance);
            //     printf("f[%i][1]: %.15f\n", i, f[i][1]);
            //     printf("f[%i][2]: %.15f\n", i, f[i][2]);
            // }
            // -------------------------------------- //
            // ------- UPDATE POSITIONS AND VELOCITIES ------- //
        	float dt = DT;
        	// update positions and velocities
        	for(i = 0; i < n; i++)
        	{
        		v[i][0] += ((f[i][0]-DAMP*v[i][0])/mass[i])*dt;
        		v[i][1] += ((f[i][1]-DAMP*v[i][1])/mass[i])*dt;
        		v[i][2] += ((f[i][2]-DAMP*v[i][2])/mass[i])*dt;

        		p[i][0] += v[i][0]*dt;
        		p[i][1] += v[i][1]*dt;
        		p[i][2] += v[i][2]*dt;
        	}
            // -------------------------------------- //
            printf("updated positions:\n");
    		for(i = 0; i < n; i++)
            {
                printf("p[%i][0]: %.15f\n", i, p[i][0]);
                printf("p[%i][1]: %.15f\n", i, p[i][1]);
                printf("p[%i][2]: %.15f\n", i, p[i][2]);
            }
            // ------- GET TOTAL KINETIC ENERGY ------- //
        	//calculate total kinetic energy
        	total_kinetic_energy = 0.0;
            // printf("total kinetic energy: %.15f\n", total_kinetic_energy);
        	for(i = 0; i < n; i++)
        	{
        		// total kinetic energy = summation{i=1}{n} 1/2 m v_i^2
        		total_kinetic_energy += 0.5*mass[i]*(v[i][0]*v[i][0] + v[i][1]*v[i][1] +v[i][2]*v[i][2]);
        	}
            // -------------------------------------- //
            test = test + 1;
            if (test < 680169) {
                printf("total kinetic energy: %.15f\n", total_kinetic_energy);
                printf("count: %i\n", test);
            }
            else {
                break;
            }
            // if (count > 5) break;
		}
        // ------- IDENTIFY SHAPE ------- //
    	float squared_distance;

    	// figure out which shape is formed
    	float total_body_to_body_distance = 0.0;
    	for(i = 0; i < n - 1; i++)
    	{
    		for(j = i + 1; j < n; j++)
    		{
    			dx = p[j][0]-p[i][0];
    			dy = p[j][1]-p[i][1];
    			dz = p[j][2]-p[i][2];
    			squared_distance = dx*dx + dy*dy + dz*dz;
    			distance = sqrt(squared_distance);
    			total_body_to_body_distance += distance;
    		}
    	}
    	// theoretical distance: 16.2426
    	if(total_body_to_body_distance < 16.5426 && 15.9426 < total_body_to_body_distance)
    	{
            OCTA_GPU[id] = 1.0;
    	}
    	// theoretical distance: 17.168
    	else if(total_body_to_body_distance < 17.468 && 16.868 < total_body_to_body_distance)
    	{
            TETRA_GPU[id] = 1.0;
    	}
    	else
    	{
            OTHER_GPU[id] = 1.0;
    	}
        // -------------------------------------- //
    }
    // TETRA_GPU[id] = 5.0;
}

int main(int argc, char** argv)
{
	srand((unsigned int)time(NULL));
    printf("NUMBER_OF_THREADS = %i\n", NUMBER_OF_THREADS);
    // int i;
    timeval start, end;

    // Set the thread structure that you will be using on the GPU
    set_up_cuda_devices();

    // Partitioning off the memory that you will be using
    allocate_memory();

    //---- SELF-ASSEMBLY ON GPU ----//
    gettimeofday(&start, NULL);
    // Copy Memory from CPU to GPU
    hipMemcpyAsync(OCTA_GPU, OCTA_CPU, NUMBER_OF_THREADS*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(TETRA_GPU, TETRA_CPU, NUMBER_OF_THREADS*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(OTHER_GPU, OTHER_CPU, NUMBER_OF_THREADS*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(RANDOM_POS_GPU, RANDOM_POS_CPU, N*3*NUMBER_OF_THREADS*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(RANDOM_VEL_GPU, RANDOM_VEL_CPU, N*3*NUMBER_OF_THREADS*sizeof(float), hipMemcpyHostToDevice);
    errorCheck("error copying to GPU");

    // Calling the Kernel (GPU) function.
    self_assemble<<<dimGrid,dimBlock>>>(OCTA_GPU, TETRA_GPU, OTHER_GPU,
                                        RANDOM_POS_GPU, RANDOM_VEL_GPU,
                                        NUMBER_OF_THREADS, N, time(NULL));
    errorCheck("error calling GPU function");

    // Copy Memory from GPU to CPU
    hipMemcpyAsync(OCTA_CPU, OCTA_GPU, NUMBER_OF_THREADS*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpyAsync(TETRA_CPU, TETRA_GPU, NUMBER_OF_THREADS*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpyAsync(OTHER_CPU, OTHER_GPU, NUMBER_OF_THREADS*sizeof(double), hipMemcpyDeviceToHost);
    errorCheck("error copying to CPU");

    // TODO: sum up counts from GPU vars then calculate frequencies for the two
    // shapes
    for (int i = 0; i < NUMBER_OF_THREADS; i++) {
        printf("TETRA_CPU[%i]: %.15f\n", i, TETRA_CPU[i]);
    }
    float octa_count = 0.0;
    for (int i = 0; i < dimGrid.x; i++) {
        octa_count += OCTA_CPU[i * dimBlock.x];
    }
    printf("octa cpu count: %.15f\n", octa_count);
    float tetra_count = 0.0;
    for (int i = 0; i < dimGrid.x; i++) {
        tetra_count += TETRA_CPU[i * dimBlock.x];
    }
    printf("tetra cpu count: %.15f\n", tetra_count);
    float other_count = 0.0;
    for (int i = 0; i < dimGrid.x; i++) {
        other_count += OTHER_CPU[i * dimBlock.x];
    }
    printf("other cpu count: %.15f\n", other_count);

	// num_experiments = (float)run_count + 1.0;
	// octa_rate = octa_count/num_experiments;
	// tetra_rate = tetra_count/num_experiments;
	// other_rate = other_count/num_experiments;
	// printf("run count: %i\t octa_rate: %.2f\t tetra_rate: %.2f\t other_rate: %.2f\n",
	// 		(run_count+1), octa_rate, tetra_rate, other_rate);

    // add first entry in each block since blocks
    // can't communicate
    // dot_result = 0.0;
    // for (int i = 0; i < dimGrid.x; i++) {
    //     dot_result += OTHER_CPU[i * dimBlock.x];
    // }
    // printf("GPU dot product: %.15f\n", dot_result);
    //
    // // Stopping the timer
    // gettimeofday(&end, NULL);
    // // Calculating the total time used in the addition and converting it to milliseconds.
    // time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
    // // Displaying the time
    // printf("Time in milliseconds= %.15f\n", (time/1000.0));
    //
    // // Displaying vector info you will want to comment out the vector print line when your
    // // vector becomes big. This is just to make sure everything is running correctly.
    // for(i = 0; i < N; i++)
    // {
    //     //printf("A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", i, OCTA_CPU[i], i, TETRA_CPU[i], i, OTHER_CPU[i]);
    // }

    // Displaying the last value of the addition for a check when all vector display has been commented out.
    //printf("Last Values are A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", N-1, OCTA_CPU[N-1], N-1, TETRA_CPU[N-1], N-1, OTHER_CPU[N-1]);

    // You're done so cleanup your mess.
    clean_up(OCTA_CPU, TETRA_CPU, OTHER_CPU, OCTA_GPU, TETRA_GPU, OTHER_GPU);

    return(0);
}

// OUTPUT
/*
Number of blocks: 8
CPU dot product: 41654167500.000000000000000
Time in milliseconds= 0.036000000000000

GPU dot product: 41654167500.000000000000000
Time in milliseconds= 0.144000000000000
*/
